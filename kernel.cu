#include "hip/hip_runtime.h"
﻿#define GLEW_STATIC
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define _USE_MATH_DEFINES
#include <cmath>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#include "stb_image.h"
#include "stb_image_write.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <chrono>
#include <ctime>
#include <iomanip>
#include <sstream>
#include <string>


#define WIDTH 800
#define HEIGHT 600
#define LANCZOS_A 3

__device__ float lanczosWeight(float x) {
    if (x == 0.0f) return 1.0f;
    if (x < -LANCZOS_A || x > LANCZOS_A) return 0.0f;
    x *= M_PI;
    return LANCZOS_A * sinf(x) * sinf(x / LANCZOS_A) / (x * x);
}

__global__ void lanczosUpscaleKernel(uchar4* output, const uchar4* input, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outputWidth && y < outputHeight) {
        float inputX = (float)x * inputWidth / outputWidth;
        float inputY = (float)y * inputHeight / outputHeight;

        int x1 = (int)floorf(inputX) - LANCZOS_A + 1;
        int y1 = (int)floorf(inputY) - LANCZOS_A + 1;

        float4 sum = make_float4(0, 0, 0, 0);
        float totalWeight = 0.0f;

        for (int j = 0; j < 2 * LANCZOS_A; ++j) {
            for (int i = 0; i < 2 * LANCZOS_A; ++i) {
                int ix = min(max(x1 + i, 0), inputWidth - 1);
                int iy = min(max(y1 + j, 0), inputHeight - 1);

                float weight = lanczosWeight(inputX - (x1 + i)) * lanczosWeight(inputY - (y1 + j));
                uchar4 pixel = input[iy * inputWidth + ix];

                sum.x += pixel.x * weight;
                sum.y += pixel.y * weight;
                sum.z += pixel.z * weight;
                sum.w += pixel.w * weight;
                totalWeight += weight;
            }
        }

        output[y * outputWidth + x] = make_uchar4(sum.x / totalWeight, sum.y / totalWeight, sum.z / totalWeight, sum.w / totalWeight);
    }
}

std::string generateFilename(const std::string& originalFilename) {
    auto now = std::chrono::system_clock::now();
    auto in_time_t = std::chrono::system_clock::to_time_t(now);

    std::stringstream ss;
    ss << std::put_time(std::localtime(&in_time_t), "%Y%m%d_%H%M");

    std::string timestamp = ss.str();
    std::string basename = originalFilename.substr(0, originalFilename.find_last_of("."));

    return basename + "_upscaled_" + timestamp + ".png";
}

int main()
{
    // Load image using stb_image
    int inputWidth, inputHeight, channels;
    std::string inputFilename = "C:/Users/lilyx/source/repos/cudaHackathon2/nature.jpg";
    unsigned char* imageData = stbi_load(inputFilename.c_str(), &inputWidth, &inputHeight, &channels, 4);
    if (!imageData) {
        fprintf(stderr, "Failed to load image\n");
        return -1;
    }

    int outputWidth = inputWidth * 5;  // 5x upscale in width
    int outputHeight = inputHeight;

    if (!glfwInit()) {
        fprintf(stderr, "Failed to initialize GLFW\n");
        return -1;
    }

    GLFWwindow* window = glfwCreateWindow(outputWidth, outputHeight, "CUDA Image Upscale", NULL, NULL);
    if (!window) {
        fprintf(stderr, "Failed to create GLFW window\n");
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK) {
        fprintf(stderr, "Failed to initialize GLEW\n");
        return -1;
    }

    GLuint textureID;
    glGenTextures(1, &textureID);
    glBindTexture(GL_TEXTURE_2D, textureID);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, outputWidth, outputHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    hipGraphicsResource* cudaTextureResource;
    hipGraphicsGLRegisterImage(&cudaTextureResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);

    uchar4* d_input = nullptr;
    uchar4* d_output = nullptr;
    hipMalloc(&d_input, inputWidth * inputHeight * sizeof(uchar4));
    hipMalloc(&d_output, outputWidth * outputHeight * sizeof(uchar4));

    hipMemcpy(d_input, imageData, inputWidth * inputHeight * sizeof(uchar4), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((outputWidth + blockSize.x - 1) / blockSize.x, (outputHeight + blockSize.y - 1) / blockSize.y);

    lanczosUpscaleKernel << <gridSize, blockSize >> > (d_output, d_input, inputWidth, inputHeight, outputWidth, outputHeight);
    hipDeviceSynchronize();

    // Save the upscaled image
    unsigned char* h_output = new unsigned char[outputWidth * outputHeight * 4];
    hipMemcpy(h_output, d_output, outputWidth * outputHeight * sizeof(uchar4), hipMemcpyDeviceToHost);

    std::string outputFilename = generateFilename(inputFilename);
    stbi_write_png(outputFilename.c_str(), outputWidth, outputHeight, 4, h_output, outputWidth * 4);
    delete[] h_output;

    while (!glfwWindowShouldClose(window)) {
        hipArray* textureArray;
        hipGraphicsMapResources(1, &cudaTextureResource, 0);
        hipGraphicsSubResourceGetMappedArray(&textureArray, cudaTextureResource, 0, 0);
        hipMemcpy2DToArray(textureArray, 0, 0, d_output, outputWidth * sizeof(uchar4), outputWidth * sizeof(uchar4), outputHeight, hipMemcpyDeviceToDevice);
        hipGraphicsUnmapResources(1, &cudaTextureResource, 0);

        glClear(GL_COLOR_BUFFER_BIT);
        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, textureID);
        glBegin(GL_QUADS);
        glTexCoord2f(0, 1); glVertex2f(-1, -1);  // Flipped vertically
        glTexCoord2f(1, 1); glVertex2f(1, -1);   // Flipped vertically
        glTexCoord2f(1, 0); glVertex2f(1, 1);    // Flipped vertically
        glTexCoord2f(0, 0); glVertex2f(-1, 1);   // Flipped vertically
        glEnd();

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipFree(d_input);
    hipFree(d_output);
    hipGraphicsUnregisterResource(cudaTextureResource);
    glDeleteTextures(1, &textureID);
    glfwDestroyWindow(window);
    glfwTerminate();
    hipDeviceReset();

    stbi_image_free(imageData);

    return 0;
}